#include "hip/hip_runtime.h"
#include <math.h>
#include <memory.h>
#include <random>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"
#include ""
#include "Neural_Networks.h"

#define NUMBER_THREADS 64

__global__ void Activate(Layer layer) {
	int j = blockIdx.y * blockDim.x + threadIdx.x;

	if (j < layer.number_nodes) {
		float *neuron = &layer.neuron[blockIdx.x * layer.number_nodes];

		neuron[j] = neuron[j] + layer.bias[j];
		neuron[j] = 1 / (1 + exp(-neuron[j]));
	}
}
__global__ void Adjust_Bias(Layer layer, double learning_rate) {
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (j < layer.number_nodes) {
		double sum = 0;

		for (int h = 0; h < layer.batch_size; h++) {
			sum += layer.error[h * layer.number_nodes + j];
		}
		layer.bias[j] -= learning_rate * sum;
	}
}
__global__ void Adjust_Weight(Layer layer, Layer parent_layer, Connection connection, double learning_rate) {
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (j < connection.number_weights) {
		double sum = 0;

		for (int k = j / parent_layer.number_nodes, l = j % parent_layer.number_nodes, h = 0; h < layer.batch_size; h++) {
			sum += layer.error[h * layer.number_nodes + k] * parent_layer.neuron[h * parent_layer.number_nodes + l];
		}
		connection.weight[j] -= learning_rate * sum;
	}
}
__global__ void Backward(Layer layer, Layer parent_layer, Connection connection) {
	int j = blockIdx.y * blockDim.x + threadIdx.x;

	if (j < parent_layer.number_nodes) {
		float *error = &layer.error[blockIdx.x * layer.number_nodes];
		float *prev_error = &parent_layer.error[blockIdx.x * parent_layer.number_nodes];

		double sum = 0;

		for (int l = 0; l < layer.number_nodes; l++) {
			sum += error[l] * connection.weight[l * parent_layer.number_nodes + j];
		}
		prev_error[j] += sum;
	}
}
__global__ void Calculate_Error(Layer layer, float y_data[]) {
	int j = blockIdx.y * blockDim.x + threadIdx.x;

	if (j < layer.number_nodes) {
		int index = blockIdx.x * layer.number_nodes + j;

		layer.error[index] = 2 * (layer.neuron[index] - y_data[index]) / (layer.batch_size * layer.number_nodes);
	}
}
__global__ void Calculate_Loss(Layer layer, float y_data[]) {
	__shared__ double sum[NUMBER_THREADS];

	sum[threadIdx.x] = 0;

	for (int j = threadIdx.x; j < layer.batch_size * layer.number_nodes; j += blockDim.x) {
		sum[threadIdx.x] += (layer.neuron[j] - y_data[j]) * (layer.neuron[j] - y_data[j]);
	}
	for (int h = (blockDim.x >> 1); h; h = (h >> 1)) {
		__syncthreads();

		if (threadIdx.x < h) {
			sum[threadIdx.x] += sum[threadIdx.x + h];
		}
	}
	if (threadIdx.x == 0) {
		y_data[0] = sum[0];
	}
}
__global__ void Differentiate(Layer layer) {
	int j = blockIdx.y * blockDim.x + threadIdx.x;

	if (j < layer.number_nodes) {
		float *error = &layer.error[blockIdx.x * layer.number_nodes];
		float *neuron = &layer.neuron[blockIdx.x * layer.number_nodes];

		error[j] *= (1 - neuron[j]) * neuron[j];
	}
}
__global__ void Forward(Layer layer, Layer parent_layer, Connection connection) {
	int j = blockIdx.y * blockDim.x + threadIdx.x;

	if (j < layer.number_nodes) {
		float *neuron = &layer.neuron[blockIdx.x * layer.number_nodes];
		float *prev_neuron = &parent_layer.neuron[blockIdx.x * parent_layer.number_nodes];

		double sum = 0;

		for (int l = 0; l < parent_layer.number_nodes; l++) {
			sum += prev_neuron[l] * connection.weight[j * parent_layer.number_nodes + l];
		}
		neuron[j] = sum;
	}
}

Connection::Connection(Layer *layer, Layer *parent_layer, double scale) {
	float *memory;

	this->layer = layer;
	this->parent_layer = parent_layer;
	this->number_weights = layer->number_nodes * parent_layer->number_nodes;

	memory = new float[number_weights];
	hipMalloc(&weight, sizeof(float) * number_weights);

	for (int i = 0; i < number_weights; i++) {
		memory[i] = scale * (2.0 * rand() / RAND_MAX - 1);
	}
	hipMemcpy(weight, memory, sizeof(float) * number_weights, hipMemcpyHostToDevice);
	delete[] memory;
}
Connection::~Connection() {}

void Connection::Destruct() {
	hipFree(weight);
}


Layer::Layer(int number_nodes) {
	this->batch_size = 1;
	this->number_nodes = number_nodes;

	hipMalloc(&bias, sizeof(float) * number_nodes);
	hipMemset(bias, 0, sizeof(float) * number_nodes);
	hipMalloc(&error, sizeof(float) * number_nodes);
	hipMalloc(&neuron, sizeof(float) * number_nodes);
}
Layer::~Layer() {}

void Layer::Destruct() {
	for (int i = 0; i < connection.size(); i++) {
		connection[i]->Destruct();
		delete connection[i];
	};
	hipFree(bias);
	hipFree(error);
	hipFree(neuron);
}
void Layer::Forward() {
	dim3 number_blocks(batch_size, number_nodes / NUMBER_THREADS + 1);

	for (int k = 0; k < connection.size(); k++) {
		::Forward << <number_blocks, NUMBER_THREADS >> > (*this, *connection[k]->parent_layer, *connection[k]);
	}
	::Activate << <number_blocks, NUMBER_THREADS >> >(*this);
}
void Layer::Resize_Memory(int batch_size) {
	this->batch_size = batch_size;

	hipFree(error);
	hipFree(neuron);
	hipMalloc(&error, sizeof(float) * batch_size * number_nodes);
	hipMalloc(&neuron, sizeof(float) * batch_size * number_nodes);
}


void Neural_Networks::Resize_Memory(int batch_size) {
	if (this->batch_size != batch_size) {
		for (int i = 0; i < layer.size(); i++) {
			layer[i]->Resize_Memory(batch_size);
		}
		this->batch_size = batch_size;
	}
}

double Neural_Networks::Calculate_Loss(Layer *layer, float **y_batch) {
	float loss, *y_data, *memory = new float[layer->batch_size * layer->number_nodes];

	hipMalloc(&y_data, sizeof(float) * layer->batch_size * layer->number_nodes);

	for (int h = 0; h < batch_size; h++) {
		memcpy(&memory[h * layer->number_nodes], y_batch[h], sizeof(float) * layer->number_nodes);
	}
	hipMemcpy(y_data, memory, sizeof(float) * layer->batch_size * layer->number_nodes, hipMemcpyHostToDevice);
	::Calculate_Loss << <1, NUMBER_THREADS >> > (*layer, y_data);
	hipMemcpy(&loss, y_data, sizeof(float), hipMemcpyDeviceToHost);

	hipFree(y_data);
	delete[] memory;

	return loss;
}

Neural_Networks::Neural_Networks() {
	batch_size = 1;
}
Neural_Networks::~Neural_Networks() {
	for (int i = 0; i < layer.size(); i++) {
		layer[i]->Destruct();
		delete layer[i];
	}
}

void Neural_Networks::Add(int number_nodes) {
	this->layer.push_back(new Layer(number_nodes));
}
void Neural_Networks::Compile(double learning_rate) {
	this->learning_rate = learning_rate;
}
void Neural_Networks::Connect(int from, int to, double scale) {
	Connection *connection = new Connection(layer[from], layer[to], scale);

	this->connection.push_back(connection);
	layer[from]->connection.push_back(connection);
}
void Neural_Networks::Predict(float input[], float output[]) {
	Predict(&input, &output);
}
void Neural_Networks::Predict(float **input, float **output, int batch_size) {
	float *memory = new float[batch_size * ((layer.front()->number_nodes > layer.back()->number_nodes) ? (layer.front()->number_nodes) : (layer.back()->number_nodes))];

	Resize_Memory(batch_size);

	for (int h = 0, i = 0; h < batch_size; h++) {
		memcpy(&memory[h * layer[i]->number_nodes], input[h], sizeof(float) * layer[i]->number_nodes);
	}
	hipMemcpy(layer.front()->neuron, memory, sizeof(float) * batch_size * layer.front()->number_nodes, hipMemcpyHostToDevice);

	for (int i = 1; i < layer.size(); i++) {
		layer[i]->Forward();
	}
	hipMemcpy(memory, layer.back()->neuron, sizeof(float) * batch_size * layer.back()->number_nodes, hipMemcpyDeviceToHost);

	for (int h = 0, i = layer.size() - 1; h < batch_size; h++) {
		memcpy(output[h], &memory[h * layer[i]->number_nodes], sizeof(float) * layer[i]->number_nodes);
	}
	delete[] memory;
}

double Neural_Networks::Evaluate(float **x_test, float **y_test, int test_size, int batch_size) {
	float **x_batch = new float*[batch_size];
	float **y_batch = new float*[batch_size];

	double loss = 0;

	for (int g = 0, h = 0; g < test_size; g++) {
		x_batch[h] = x_test[g];
		y_batch[h] = y_test[g];

		if (++h == batch_size || g == test_size - 1) {
			float *memory = new float[h * layer[0]->number_nodes];

			Resize_Memory(h);

			// copy x_test to neuron
			while (--h >= 0) {
				memcpy(&memory[h * layer[0]->number_nodes], x_batch[h], sizeof(float) * layer[0]->number_nodes);
			}
			hipMemcpy(layer[0]->neuron, memory, sizeof(float) * this->batch_size * layer[0]->number_nodes, hipMemcpyHostToDevice);
			delete[] memory;
			h = 0;

			// forward propagation
			for (int i = 1; i < layer.size(); i++) {
				layer[i]->Forward();
			}

			// calculate loss
			loss += Calculate_Loss(layer.back(), y_batch);
		}
	}
	delete[] x_batch;
	delete[] y_batch;

	return loss / (test_size * layer.back()->number_nodes);
}
double Neural_Networks::Fit(float **x_train, float **y_train, int train_size, int batch_size) {
	float **x_batch = new float*[batch_size];
	float **y_batch = new float*[batch_size];

	double loss = 0;

	for (int g = 0, h = 0; g < train_size; g++) {
		x_batch[h] = x_train[g];
		y_batch[h] = y_train[g];

		if (++h == batch_size || g == train_size - 1) {
			float *memory = new float[h * layer[0]->number_nodes];

			Resize_Memory(h);

			// copy x_train to neuron
			while (--h >= 0) {
				memcpy(&memory[h * layer[0]->number_nodes], x_batch[h], sizeof(float) * layer[0]->number_nodes);
			}
			hipMemcpy(layer[0]->neuron, memory, sizeof(float) * this->batch_size * layer[0]->number_nodes, hipMemcpyHostToDevice);
			delete[] memory;
			h = 0;

			// initialize error to zero
			for (int i = 0; i < layer.size(); i++) {
				hipMemset(layer[i]->error, 0, sizeof(float) * layer[i]->batch_size * layer[i]->number_nodes);
			}

			// forward propagation
			for (int i = 1; i < layer.size(); i++) {
				layer[i]->Forward();
			}

			// calculate loss
			loss += Calculate_Loss(layer.back(), y_batch);

			// error backpropagation
			for (int i = layer.size() - 1; i > 0; i--) {
				Layer *layer = this->layer[i];

				if (i == this->layer.size() - 1) {
					dim3 number_blocks(layer->batch_size, layer->number_nodes / NUMBER_THREADS + 1);

					float *y_data, *memory = new float[layer->batch_size * layer->number_nodes];

					hipMalloc(&y_data, sizeof(float) * layer->batch_size * layer->number_nodes);

					for (int h = 0; h < layer->batch_size; h++) {
						memcpy(&memory[h * layer->number_nodes], y_batch[h], sizeof(float) * layer->number_nodes);
					}
					hipMemcpy(y_data, memory, sizeof(float) * layer->batch_size * layer->number_nodes, hipMemcpyHostToDevice);
					::Calculate_Error << <number_blocks, NUMBER_THREADS >> > (*layer, y_data);

					hipFree(y_data);
					delete[] memory;
				}
				else {
					// backpropagate error
					for (int k = 0; k < layer->connection.size(); k++) {
						Connection *connection = layer->connection[k];

						dim3 number_blocks(connection->parent_layer->batch_size, connection->parent_layer->number_nodes / NUMBER_THREADS + 1);

						::Backward << <number_blocks, NUMBER_THREADS >> > (*layer, *connection->parent_layer, *connection);
					}
				}

				dim3 number_blocks(layer->batch_size, layer->number_nodes / NUMBER_THREADS + 1);

				::Differentiate << <number_blocks, NUMBER_THREADS >> > (*layer);
			}

			// adjust bias
			for (int i = 0; i < layer.size(); i++) {
				::Adjust_Bias << <layer[i]->number_nodes / NUMBER_THREADS + 1, NUMBER_THREADS >> > (*layer[i], learning_rate);
			}

			// adjust weight
			for (int i = 0; i < connection.size(); i++) {
				Connection *connection = this->connection[i];

				::Adjust_Weight << <connection->number_weights / NUMBER_THREADS + 1, NUMBER_THREADS >> > (*connection->layer, *connection->parent_layer, *connection, learning_rate);
			}
		}
	}
	delete[] x_batch;
	delete[] y_batch;

	return loss / (train_size * layer.back()->number_nodes);
}
